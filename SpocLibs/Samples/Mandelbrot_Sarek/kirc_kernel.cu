
#include <hip/hip_runtime.h>
__device__ float spoc_fadd ( float a, float b ) { return (a + b);}
__device__ float spoc_fminus ( float a, float b ) { return (a - b);}
__device__ float spoc_fmul ( float a, float b ) { return (a * b);}
__device__ float spoc_fdiv ( float a, float b ) { return (a / b);}
#ifdef __cplusplus
extern "C" {
#endif

__global__ void spoc_dummy ( int* spoc_var0, int spoc_var1, int spoc_var2, float spoc_var3 ) {
{int spoc_var5;
{int spoc_var6;
{int spoc_var7;
{int spoc_var8;
{int spoc_var9;
{float spoc_var10;
{float spoc_var11;
{float spoc_var12;
{float spoc_var13;
{float spoc_var14;
{float spoc_var15;
{float spoc_var16;
spoc_var5 = (threadIdx.y + (blockIdx.y * blockDim.y)); ;
spoc_var6 = (threadIdx.x + (blockIdx.x * blockDim.x)); ;
if (spoc_var5 >= 1000 || spoc_var6 >= 1000){
  return  ;}
 ;
spoc_var7 = (spoc_var6 + spoc_var1); ;
spoc_var8 = (spoc_var5 + spoc_var2); ;
spoc_var9 = 0; ;
spoc_var10 = 0.f; ;
spoc_var11 = 0.f; ;
spoc_var12 = 0.f; ;
spoc_var13 = 0.f; ;
spoc_var14 = ((4.f * (((float) (spoc_var7)  / (float) (1000) ) / spoc_var3)) - 2.f); ;
spoc_var15 = ((4.f * (((float) (spoc_var8)  / (float) (1000) ) / spoc_var3)) - 2.f); ;
spoc_var16 = ((spoc_var10 * spoc_var10) + (spoc_var11 * spoc_var11)); ;
while (spoc_var9 < 50 && spoc_var16 <= 4.f){
  spoc_var9 = (spoc_var9 + 1); ;
  spoc_var12 = (((spoc_var10 * spoc_var10) - (spoc_var11 * spoc_var11)) + spoc_var14); ;
  spoc_var13 = (((2.f * spoc_var10) * spoc_var11) + spoc_var15); ;
  spoc_var10 = spoc_var12; ;
  spoc_var11 = spoc_var13; ;
  spoc_var16 = ((spoc_var10 * spoc_var10) + (spoc_var11 * spoc_var11));} ;
spoc_var0[((spoc_var5 * 1000) + spoc_var6)] = spoc_var9;;
}
;
}
;
}
;
}
;
}
;
}
;
}
;
}
;
}
;
}
;
}
;
}


}
#ifdef __cplusplus
}
#endif